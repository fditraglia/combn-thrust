// .cu
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <stdlib.h>
#include <thrust/transform.h>
#include <stdio.h>
#include <boost/math/special_functions/binomial.hpp>

using namespace std;

__device__
int next_comb(int *comb, int m, int n){
    printf("Inside next_comb\n");

    int i = m - 1;
    ++comb[i];
    
    while((i >= 0) && (comb[i] >= n - m + 1 - i)){
        --i;
        ++comb[i];
    }
    if(comb[0] == 1){
        return 0;
    }
    for(i = i + 1; i < m; ++i){
        comb[i] = comb[i-1] + 1;
    }
    return 1;
}

__device__
void find_comb(int idx, int *x, int m, int n, int*pos){
    printf("Inside find_comb");

    // test pos passed correctly
     for(int i = 0; i < n-m+1; i++){
        printf("Dpos [%d] = %d \n",i,pos[i]);
    } 

    //printf("%d, %d\n", i, x[i]);
    //printf("m = %d\n", m);
    int *comb = new int[m];
    for(int i = 0; i < m; i++){
        comb[i] = i;
    }
    int new_n= n - idx;
    
    /*
     printf("the x inside = ");
     for(int i = 0; i < n; i++){
     printf("%d ", x[i]);
     }
     printf("\n");
     */
    //	printf("	%d %d %d\n", comb[0], comb[1], comb[2]);
    printf("index %d has n = %d\n", idx, new_n);
    
    printf("ANSWER %d %d %d\n", x[comb[0] + idx], x[comb[1]+idx], x[comb[2]+idx]);
    printf("after 1st one %d %d %d\n", comb[0], comb[1], comb[2]);
    
    //	while(next_comb(comb, m, new_n)){
    //		printf("inside whiel?");
    //		printf("%d %d %d\n", x[comb[0]], x[comb[1]], x[comb[2]]);
    //		printf("		%d %d %d\n", comb[0], comb[1], comb[2]);
    //	}
    
    while(true){
        printf(" inside comb is: %d %d %d, i = nothing\n", comb[0], comb[1], comb[2]);
        int i = m - 1;
        ++comb[i];
        
        printf("	after ++comb, comb is %d %d %d, i = %d\n", comb[0], comb[1], comb[2], i);
        while((i >= 0) && (comb[i] >= new_n - m + 1 + i)){
            --i;
            ++comb[i];
        }
        printf("	after while, comb is %d %d %d, i = %d\n", comb[0], comb[1], comb[2], i);
        if(comb[0] == 1){
            break;
        }
        printf("	after if, comb is %d %d %d, i = %d\n", comb[0], comb[1], comb[2], i);
        for(i = i + 1; i < m; ++i){
            comb[i] = comb[i-1] + 1;
        }
        printf("	after for, comb is %d %d %d, i = %d\n", comb[0], comb[1], comb[2], i);
        //return 1;
        
        printf("ANSWER %d %d %d\n", x[comb[0]+idx], x[comb[1]+idx], x[comb[2]+idx]);
    }
}


struct comb {
    
    const thrust::device_vector<int>::iterator x;
    const thrust::device_vector<int>::iterator r;//might use this to store result?
    const thrust::device_vector<int>::iterator pos; 
    int n;
    int m;
    int *x_ptr, *r_ptr, *pos_ptr;
    int *comb_arr;
    
    comb(thrust::device_vector<int>::iterator _x, thrust::device_vector<int>::iterator _r, int _n, int _m,thrust::device_vector<int>::iterator _pos):
    x(_x),
    r(_r),
    n(_n),
    m(_m),
    pos(_pos)
    {
        x_ptr = thrust::raw_pointer_cast(&x[0]);
        r_ptr = thrust::raw_pointer_cast(&r[0]);
        pos_ptr = thrust::raw_pointer_cast(&pos[0]);
    }
    
    __device__
    void operator()(int i)
    {
        if(i <= n - m)
            //	printf("%d ", i);
            find_comb(i, x_ptr, m, n,pos_ptr);
    }
};

void combn(int*x, int n, int m, int *comb_arr, int *result, int nCm, int*pos ){
    //void combn(int *x, int n, int m, vector<int> result){
    
    thrust::device_vector<int> d_x(x, x+n);
    thrust::device_vector<int> d_r(result, result + (nCm * m));
    thrust::device_vector<int> d_pos(pos, pos + (n-m+1));
    
    /*
     thrust::device_vector<int> d_c(comb_arr, comb_arr + m);
     for(int i = 0; i < m; i++){
     printf("%d %d %d", comb_arr[0], comb_arr[1], comb_arr[2]);
     }
     */
    
    thrust::counting_iterator<int> begin(0);
    thrust::counting_iterator<int> end = begin + n;
    
    
    //	thrust::transform(begin, end, d_r.begin(), comb(d_x.begin(), d_r.begin(), n, m));
    thrust::for_each(begin, end, comb(d_x.begin(), d_r.begin(), n, m, d_pos.begin()));
    
    //thrust::copy(d_r.begin(), d_r.end(), result);
}



int main(){
    int n = 5;
    int m = 3;
    
    int x[n];
    int *result;
    
    // Count the number of possible combinations 
    int nCm = boost::math::binomial_coefficient<double>(n, m);

    // Size of output array 
    result = new int[nCm * m];
    
    // keeps track of the position in output
    int *pos = new int[n-m+1];	
    
    // test input array 
    cout << "x = ";
    for(int i=0; i<n; i++)
    {
        //x[i] = rand() % 5;
        x[i] = i; 
        cout << x[i] << " ";
    }
    cout << endl;
    
    int *comb_arr = new int[m];
    for(int i = 0; i < m; i++){
        comb_arr[i] = i;
    }
    

    // Calculate combination possibilities for each element in the list that 
    // start with the element in the 0th index 
    int tmp_n = n; // Why do we need a tmp_n ??
    int k = 0;
    for(int i = 0; i < (n-m+1); i++){
        pos[i] = boost::math::binomial_coefficient<double>(tmp_n - i - 1, m-1);
        k++;
    }
     
    // Print for testing purposees 
    cout << "nCm = " << nCm << endl;
    cout << "k = " << k << endl;
    cout << "n-m+1 = " << n-m+1 << endl;
    for(int i = 0; i < n-m+1; i++){
        cout << "pos[" << i << "] = " << pos[i] << endl;
    } 
    
  
    //combn(x, n, m, comb_arr, result, nCm);
    combn(x, n, m, comb_arr, result, nCm,pos);
    
    cout << "result = ";
    for(int i = 0; i < n; i++){
        cout << result[i] << " ";
    }
    cout << endl;
    return 0;
    
    
}
